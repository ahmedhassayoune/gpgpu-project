#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <cstdio>
#include <thread>

#define BLOCK_SIZE 16

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err,
           const char* const func,
           const char* const file,
           const int line)
{
  if (err != hipSuccess)
    {
      std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
      std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
      // We don't exit when we encounter CUDA errors in this example.
      std::exit(EXIT_FAILURE);
    }
}

template <typename T>
__device__ inline T* eltPtr(T* baseAddress, int col, int row, size_t pitch)
{
  return (T*)((char*)baseAddress + row * pitch + col * sizeof(T)); // FIXME
}

__device__ bool hysteresis_has_changed;

//******************************************************
//**                                                  **
//**               Background Estimation              **
//**                                                  **
//******************************************************

#define _BE_FSIGN                                                              \
  std::byte **buffers, size_t *bpitches, int buffers_amount, std::byte *out,   \
    size_t opitch, int width, int height

__global__ void estimate_background_mean(_BE_FSIGN)
{
#define _BACKGROUND_ESTIMATION_MEAN_SPST // single position single thread

  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;

#ifdef _BACKGROUND_ESTIMATION_MEAN_SPST
  // compute sum per channel
  int sums[N_CHANNELS] = {0};
  std::byte* ptr;
  for (int ii = 0; ii < buffers_amount; ++ii)
    {
      ptr = buffers[ii] + yy * bpitches[ii] + xx * PIXEL_STRIDE;
      for (int jj = 0; jj < N_CHANNELS; ++jj)
        sums[jj] += (int)ptr[jj];
    }

  // compute mean per channel
  ptr = out + yy * opitch + xx * PIXEL_STRIDE;
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    ptr[ii] = (std::byte)(sums[ii] / buffers_amount);
#else
#endif

#undef _BACKGROUND_ESTIMATION_MEAN_SPST
}

__device__ void _insertion_sort(std::byte* arr, int start, int end, int step)
{
  for (int ii = start + step; ii < end; ii += step)
    {
      int jj = ii;

      while (jj > start && arr[jj - step] > arr[jj])
        {
          std::byte tmp = arr[jj - step];
          arr[jj - step] = arr[jj];
          arr[jj] = tmp;
          jj -= step;
        }
    }
}

__global__ void estimate_background_median(_BE_FSIGN)
{
#define _BACKGROUND_ESTIMATION_MEDIAN_SPST // single position single thread

  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;

#ifdef _BACKGROUND_ESTIMATION_MEDIAN_SPST
  // 3 channels, at most 42 buffers
  // 4 channels, at most 32 buffers
  std::byte B[128];

  // for each buffer, store pixel at (yy, xx)
  for (int ii = 0; ii < buffers_amount; ++ii)
    {
      std::byte* ptr = buffers[ii] + yy * bpitches[ii] + xx * PIXEL_STRIDE;
      int jj = ii * N_CHANNELS;
      for (int kk = 0; kk < N_CHANNELS; ++kk)
        B[jj + kk] = ptr[kk];
    }

  // the median is computed for each channel
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    _insertion_sort(B, ii, buffers_amount * N_CHANNELS, N_CHANNELS);

  // select mid
  // not treating differently even and odd `buffers_amount`
  // in order to avoid if clause inside a kernel
  std::byte* ptr = out + yy * opitch + xx * PIXEL_STRIDE;
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    ptr[ii] = B[(buffers_amount / 2) * N_CHANNELS + ii];
#else
#endif

#undef _BACKGROUND_ESTIMATION_MEDIAN_SPST
}

#undef _BE_FSIGN

//******************************************************
//**                                                  **
//**           Conversion from RGB to LAB (GPU)       **
//**                                                  **
//******************************************************

__device__ void
rgbToXyz(float r, float g, float b, float& x, float& y, float& z)
{
  const float D65_XYZ[9] = {0.412453f, 0.357580f, 0.180423f,
                            0.212671f, 0.715160f, 0.072169f,
                            0.019334f, 0.119193f, 0.950227f};

  r = r / 255.0f;
  g = g / 255.0f;
  b = b / 255.0f;

#define GAMMA_CORRECT(C)                                                       \
  ((C) > 0.04045f ? powf(((C) + 0.055f) / 1.055f, 2.4f) : (C) / 12.92f)
  r = GAMMA_CORRECT(r);
  g = GAMMA_CORRECT(g);
  b = GAMMA_CORRECT(b);
#undef GAMMA_CORRECT

  x = r * D65_XYZ[0] + g * D65_XYZ[1] + b * D65_XYZ[2];
  y = r * D65_XYZ[3] + g * D65_XYZ[4] + b * D65_XYZ[5];
  z = r * D65_XYZ[6] + g * D65_XYZ[7] + b * D65_XYZ[8];
}

__device__ void
xyzToLab(float x, float y, float z, float& l, float& a, float& b)
{
  const float D65_Xn = 0.95047f;
  const float D65_Yn = 1.00000f;
  const float D65_Zn = 1.08883f;

  x /= D65_Xn;
  y /= D65_Yn;
  z /= D65_Zn;

  const float epsilon = 0.008856f;
  const float kappa = 903.3f;

#define NONLINEAR(C)                                                           \
  ((C) > epsilon ? powf((C), 1.0f / 3.0f) : ((kappa * (C) + 16.0f) / 116.0f))
  float fx = NONLINEAR(x);
  float fy = NONLINEAR(y);
  float fz = NONLINEAR(z);
#undef NONLINEAR

  l = (116.0f * fy) - 16.0f;
  a = 500.0f * (fx - fy);
  b = 200.0f * (fy - fz);
}

__global__ void rgbToLabDistanceKernel(std::byte* referenceBuffer,
                                       size_t rpitch,
                                       std::byte* buffer,
                                       size_t bpitch,
                                       float* distanceArray,
                                       size_t dpitch,
                                       const int width,
                                       const int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  rgb* lineptrReference = (rgb*)(referenceBuffer + y * rpitch);
  float rRef = lineptrReference[x].r;
  float gRef = lineptrReference[x].g;
  float bRef = lineptrReference[x].b;

  float XRef, YRef, ZRef;
  rgbToXyz(rRef, gRef, bRef, XRef, YRef, ZRef);

  float LRef, ARef, BRef;
  xyzToLab(XRef, YRef, ZRef, LRef, ARef, BRef);

  LAB referenceLab = {LRef, ARef, BRef};

  rgb* lineptr = (rgb*)(buffer + y * bpitch);
  float r = lineptr[x].r;
  float g = lineptr[x].g;
  float b = lineptr[x].b;

  float X, Y, Z;
  rgbToXyz(r, g, b, X, Y, Z);

  float L, A, B;
  xyzToLab(X, Y, Z, L, A, B);

  LAB currentLab = {L, A, B};
#define LAB_DISTANCE(lab1, lab2)                                               \
  (sqrtf(powf((lab1).l - (lab2).l, 2) + powf((lab1).a - (lab2).a, 2)           \
         + powf((lab1).b - (lab2).b, 2)))
  float distance = LAB_DISTANCE(currentLab, referenceLab);
#undef LAB_DISTANCE
  float* distancePtr = (float*)((char*)distanceArray + y * dpitch);
  distancePtr[x] = distance;
}

__global__ void normalizeAndConvertTo8bitKernel(std::byte* buffer,
                                                size_t bpitch,
                                                float* distanceArray,
                                                size_t dpitch,
                                                float max_distance,
                                                const int width,
                                                const int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  float* distancePtr = (float*)((char*)distanceArray + y * dpitch);
  float distance = distancePtr[x];

  uint8_t distance8bit =
    static_cast<uint8_t>(fminf(distance / max_distance * 255.0f, 255.0f));

  rgb* lineptr = (rgb*)(buffer + y * bpitch);
  lineptr[x].r = distance8bit;
  lineptr[x].g = distance8bit;
  lineptr[x].b = distance8bit;
}

//******************************************************
//**                                                  **
//**             Morphological Opening                **
//**                                                  **
//******************************************************

__global__ void morphological_erosion(std::byte* buffer,
                                      size_t bpitch,
                                      std::byte* output_buffer,
                                      size_t opitch,
                                      const int width,
                                      const int height)
{
  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;
  unsigned int min_red = 0xff;
  unsigned int min_green = 0xff;
  unsigned int min_blue = 0xff;

  // Compute the minimum value in the 5x5 neighborhood
  for (int j = yy - 2; j <= yy + 2; j++)
    {
      for (int i = xx - 2; i <= xx + 2; i++)
        {
          if (i >= 0 && i < width && j >= 0 && j < height)
            {
              min_red =
                min(min_red, (unsigned int)buffer[j * bpitch + i * N_CHANNELS]);
              min_green =
                min(min_green,
                    (unsigned int)buffer[j * bpitch + i * N_CHANNELS + 1]);
              min_blue =
                min(min_blue,
                    (unsigned int)buffer[j * bpitch + i * N_CHANNELS + 2]);
            }
        }
    }

  // Compute the minimum value in the extremities
  if (xx - 3 >= 0)
    {
      int i = xx - 3;
      min_red =
        min(min_red, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS]);
      min_green =
        min(min_green, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 1]);
      min_blue =
        min(min_blue, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 2]);
    }
  if (xx + 3 < width)
    {
      int i = xx + 3;
      min_red =
        min(min_red, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS]);
      min_green =
        min(min_green, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 1]);
      min_blue =
        min(min_blue, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 2]);
    }
  if (yy - 3 >= 0)
    {
      int j = yy - 3;
      min_red =
        min(min_red, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS]);
      min_green =
        min(min_green, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 1]);
      min_blue =
        min(min_blue, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 2]);
    }
  if (yy + 3 < height)
    {
      int j = yy + 3;
      min_red =
        min(min_red, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS]);
      min_green =
        min(min_green, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 1]);
      min_blue =
        min(min_blue, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 2]);
    }

  size_t out_index = yy * opitch + xx * N_CHANNELS;
  output_buffer[out_index] = (std::byte)min_red;
  output_buffer[out_index + 1] = (std::byte)min_green;
  output_buffer[out_index + 2] = (std::byte)min_blue;
}

__global__ void morphological_dilation(std::byte* buffer,
                                       size_t bpitch,
                                       std::byte* output_buffer,
                                       size_t opitch,
                                       const int width,
                                       const int height)
{
  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;

  unsigned int max_red = 0;
  unsigned int max_green = 0;
  unsigned int max_blue = 0;

  // Compute the maximum value in the 5x5 neighborhood
  for (int j = yy - 2; j <= yy + 2; j++)
    {
      for (int i = xx - 2; i <= xx + 2; i++)
        {
          if (i >= 0 && i < width && j >= 0 && j < height)
            {
              max_red =
                max(max_red, (unsigned int)buffer[j * bpitch + i * N_CHANNELS]);
              max_green =
                max(max_green,
                    (unsigned int)buffer[j * bpitch + i * N_CHANNELS + 1]);
              max_blue =
                max(max_blue,
                    (unsigned int)buffer[j * bpitch + i * N_CHANNELS + 2]);
            }
        }
    }

  // Compute the maximum value in the extremities
  if (xx - 3 >= 0)
    {
      int i = xx - 3;
      max_red =
        max(max_red, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS]);
      max_green =
        max(max_green, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 1]);
      max_blue =
        max(max_blue, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 2]);
    }
  if (xx + 3 < width)
    {
      int i = xx + 3;
      max_red =
        max(max_red, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS]);
      max_green =
        max(max_green, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 1]);
      max_blue =
        max(max_blue, (unsigned int)buffer[yy * bpitch + i * N_CHANNELS + 2]);
    }
  if (yy - 3 >= 0)
    {
      int j = yy - 3;
      max_red =
        max(max_red, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS]);
      max_green =
        max(max_green, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 1]);
      max_blue =
        max(max_blue, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 2]);
    }
  if (yy + 3 < height)
    {
      int j = yy + 3;
      max_red =
        max(max_red, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS]);
      max_green =
        max(max_green, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 1]);
      max_blue =
        max(max_blue, (unsigned int)buffer[j * bpitch + xx * N_CHANNELS + 2]);
    }

  int out_index = yy * opitch + xx * N_CHANNELS;
  output_buffer[out_index] = (std::byte)max_red;
  output_buffer[out_index + 1] = (std::byte)max_green;
  output_buffer[out_index + 2] = (std::byte)max_blue;
}

//******************************************************
//**                                                  **
//**               Hysteresis Threshold               **
//**                                                  **
//******************************************************

/// @brief Apply a threshold on the buffer and store the result in the marker
/// @param buffer The input buffer
/// @param bpitch The pitch of the input buffer
/// @param marker The marker buffer
/// @param mpitch The pitch of the marker buffer
/// @param width The width of the image
/// @param height The height of the image
/// @param high_threshold The high threshold
/// @return
__global__ void apply_threshold_on_marker(std::byte* buffer,
                                          size_t bpitch,
                                          bool* marker,
                                          size_t mpitch,
                                          const int width,
                                          const int height,
                                          int high_threshold)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= width || y >= height)
    return;

  rgb* buffer_line = (rgb*)(buffer + y * bpitch);
  bool* marker_line = (bool*)((std::byte*)marker + y * mpitch);

  marker_line[x] = buffer_line[x].r > high_threshold;
}

/// @brief Reconstruct the hysteresis thresholding image from the marker
/// @param buffer The input buffer
/// @param bpitch The pitch of the input buffer
/// @param out The output buffer
/// @param opitch The pitch of the output buffer
/// @param marker The marker buffer
/// @param mpitch The pitch of the marker buffer
/// @param buffer_info The buffer info
/// @param low_threshold The low threshold
/// @return
__global__ void reconstruct_image(std::byte* buffer,
                                  size_t bpitch,
                                  std::byte* out,
                                  size_t opitch,
                                  bool* marker,
                                  size_t mpitch,
                                  const int width,
                                  const int height,
                                  int low_threshold)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= width || y >= height)
    return;

  rgb* out_line = (rgb*)(out + y * opitch);
  bool* marker_line = (bool*)((std::byte*)marker + y * mpitch);

  if (!marker_line[x] || out_line[x].r != 0)
    {
      return;
    }

  // Set the pixel to white
  out_line[x].r = 255;
  out_line[x].g = 255;
  out_line[x].b = 255;

  // Mark the 8-connected neighbors if they are above the low threshold
  for (int i = -1; i <= 1; i++)
    {
      for (int j = -1; j <= 1; j++)
        {
          int ny = y + j;
          int nx = x + i;
          // Skip the current pixel
          if ((i == 0 && j == 0) || nx < 0 || nx >= width || ny < 0
              || ny >= height)
            {
              continue;
            }

          // Check if the pixel is within the image boundaries
          rgb* buffer_line = (rgb*)(buffer + ny * bpitch);
          bool* neighbor_marker_line =
            (bool*)((std::byte*)marker + ny * mpitch);
          if (!neighbor_marker_line[nx] && buffer_line[nx].r > low_threshold)
            {
              neighbor_marker_line[nx] = true;
              hysteresis_has_changed = true;
            }
        }
    }
}

//******************************************************
//**                                                  **
//**                Apply Masking                     **
//**                                                  **
//******************************************************

__global__ void apply_masking(std::byte* buffer,
                              size_t bpitch,
                              std::byte* mask,
                              size_t mpitch,
                              int width,
                              int height)
{
  int xx = blockIdx.x * blockDim.x + threadIdx.x;
  int yy = blockIdx.y * blockDim.y + threadIdx.y;

  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;
  rgb* ipptr = (rgb*)(buffer + yy * bpitch + xx * PIXEL_STRIDE);
  rgb* mpptr = (rgb*)(mask + yy * mpitch + xx * PIXEL_STRIDE);
  int red = (int)ipptr->r + (mpptr->r > 0) * ipptr->r / 2;
  ipptr->r = (uint8_t)(red > 0xff ? 0xff : red);
}

__global__ void copy_buffer_kernel(std::byte* dbuffer,
                                   size_t bpitch,
                                   std::byte* cpy_buffer,
                                   size_t cpitch,
                                   std::byte* mask,
                                   size_t mpitch,
                                   std::byte* fallback_dbuffer,
                                   size_t fpitch,
                                   const int width,
                                   const int height)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height)
    return;

  rgb* lineptr = (rgb*)(dbuffer + y * bpitch);
  rgb* cpy_lineptr = (rgb*)(cpy_buffer + y * cpitch);
  rgb* fallback_lineptr = (rgb*)(fallback_dbuffer + y * fpitch);
  rgb* mask_lineptr = (rgb*)(mask + y * mpitch);

  if (mask_lineptr[x].r == 0)
    {
      cpy_lineptr[x].r = lineptr[x].r;
      cpy_lineptr[x].g = lineptr[x].g;
      cpy_lineptr[x].b = lineptr[x].b;
    }
  else
    {
      cpy_lineptr[x].r = fallback_lineptr[x].r;
      cpy_lineptr[x].g = fallback_lineptr[x].g;
      cpy_lineptr[x].b = fallback_lineptr[x].b;
    }
}

namespace
{
  //******************************************************
  //**                                                  **
  //**           Conversion from RGB to LAB (GPU)       **
  //**                                                  **
  //******************************************************

  void rgb_to_lab_cuda(std::byte* referenceBuffer,
                       size_t rpitch,
                       std::byte* buffer,
                       size_t bpitch,
                       const frame_info* buffer_info)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;

    hipError_t err;
    float* distanceArray;
    size_t dpitch;

    err =
      hipMallocPitch(&distanceArray, &dpitch, width * sizeof(float), height);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    rgbToLabDistanceKernel<<<gridSize, blockSize>>>(
      referenceBuffer, rpitch, buffer, bpitch, distanceArray, dpitch, width,
      height);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    float* h_distanceArray = new float[width * height];
    err = hipMemcpy2D(h_distanceArray, width * sizeof(float), distanceArray,
                       dpitch, width * sizeof(float), height,
                       hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err);

    float maxDistance = 0.0f;
    for (int i = 0; i < width * height; ++i)
      {
        maxDistance = fmaxf(maxDistance, h_distanceArray[i]);
      }
    delete[] h_distanceArray;

    normalizeAndConvertTo8bitKernel<<<gridSize, blockSize>>>(
      buffer, bpitch, distanceArray, dpitch, maxDistance, width, height);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    err = hipFree(distanceArray);
    CHECK_CUDA_ERROR(err);
  }

  //******************************************************
  //**                                                  **
  //**             Morphological Opening                **
  //**                                                  **
  //******************************************************

  void opening_impl_inplace(std::byte* buffer,
                            size_t bpitch,
                            const frame_info* buffer_info)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;

    std::byte* gpu_image;
    size_t gpu_pitch;
    hipError_t err =
      hipMallocPitch(&gpu_image, &gpu_pitch, width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);

    err = hipMemcpy2D(gpu_image, gpu_pitch, buffer, bpitch, width * N_CHANNELS,
                       height, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err);

    std::byte* gpu_intermediate_image;
    size_t gpu_intermediate_pitch;
    err = hipMallocPitch(&gpu_intermediate_image, &gpu_intermediate_pitch,
                          width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);

    morphological_erosion<<<gridSize, blockSize>>>(
      gpu_image, gpu_pitch, gpu_intermediate_image, gpu_intermediate_pitch,
      width, height);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    morphological_dilation<<<gridSize, blockSize>>>(
      gpu_intermediate_image, gpu_intermediate_pitch, gpu_image, gpu_pitch,
      width, height);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    err = hipMemcpy2D(buffer, bpitch, gpu_image, gpu_pitch, width * N_CHANNELS,
                       height, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err);

    hipFree(gpu_image);
    hipFree(gpu_intermediate_image);
  }

  //******************************************************
  //**                                                  **
  //**               Hysteresis Threshold               **
  //**                                                  **
  //******************************************************

  /// @brief Apply hysteresis thresholding on the buffer
  /// @param buffer The input buffer
  /// @param bpitch The pitch of the input buffer
  /// @param buffer_info The buffer info
  /// @param low_threshold The low threshold
  /// @param high_threshold The high threshold
  void apply_hysteresis_threshold(std::byte* buffer,
                                  size_t bpitch,
                                  const frame_info* buffer_info,
                                  int low_threshold,
                                  int high_threshold)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;

    // Ensure low threshold is less than high threshold
    if (low_threshold > high_threshold)
      {
        low_threshold = high_threshold;
      }

    // Create a marker buffer to store the pixels that are above the high threshold
    bool* marker;
    size_t mpitch;
    hipError_t err;
    err = hipMallocPitch(&marker, &mpitch, width * sizeof(bool), height);
    CHECK_CUDA_ERROR(err);

    // And set it to false
    err = hipMemset2D(marker, mpitch, 0, width * sizeof(bool), height);
    CHECK_CUDA_ERROR(err);

    // Create an out buffer to store the final image
    std::byte* out_buffer;
    size_t opitch;
    err = hipMallocPitch(&out_buffer, &opitch, width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);

    // And set it to black
    err = hipMemset2D(out_buffer, opitch, 0, width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);
    apply_threshold_on_marker<<<gridSize, blockSize>>>(
      buffer, bpitch, marker, mpitch, width, height, high_threshold);

    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    // Apply hysteresis thresholding
    bool h_hysteresis_has_changed = true;
    while (h_hysteresis_has_changed)
      {
        // Copy the value of hysteresis_has_changed to device
        h_hysteresis_has_changed = false;
        err =
          hipMemcpyToSymbol(HIP_SYMBOL(hysteresis_has_changed), &h_hysteresis_has_changed,
                             sizeof(h_hysteresis_has_changed));
        CHECK_CUDA_ERROR(err);

        reconstruct_image<<<gridSize, blockSize>>>(
          buffer, bpitch, out_buffer, opitch, marker, mpitch, width, height,
          low_threshold);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);

        // Retrieve the value of hysteresis_has_changed from device
        err = hipMemcpyFromSymbol(&h_hysteresis_has_changed,
                                   HIP_SYMBOL(hysteresis_has_changed),
                                   sizeof(hysteresis_has_changed));
        CHECK_CUDA_ERROR(err);
      }

    // Copy the final image to the buffer
    err = hipMemcpy2D(buffer, bpitch, out_buffer, opitch, width * N_CHANNELS,
                       height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    hipFree(marker);
    hipFree(out_buffer);
  }

  //******************************************************
  //**                                                  **
  //**             Background Model Update              **
  //**                                                  **
  //******************************************************

  void copy_buffer(std::byte* dbuffer,
                   size_t bpitch,
                   std::byte** cpy_dbuffer,
                   size_t* cpitch,
                   std::byte* dmask,
                   size_t mpitch,
                   std::byte* fallback_dbuffer,
                   size_t fpitch,
                   const frame_info* buffer_info)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;

    hipError_t err;

    if (*cpy_dbuffer == nullptr)
      {
        // Allocate memory for the copy buffer
        err = hipMallocPitch(cpy_dbuffer, cpitch, width * N_CHANNELS, height);
        CHECK_CUDA_ERROR(err);
      }

    if (dmask == nullptr || fallback_dbuffer == nullptr)
      {
        // Copy dbuffer to cpy_buffer
        err =
          hipMemcpy2D(*cpy_dbuffer, *cpitch, dbuffer, bpitch,
                       width * N_CHANNELS, height, hipMemcpyDeviceToDevice);
        CHECK_CUDA_ERROR(err);
      }
    else
      {
        // Copy dbuffer where mask is false and fallback_dbuffer where mask is true
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                      (height + blockSize.y - 1) / blockSize.y);
        copy_buffer_kernel<<<gridSize, blockSize>>>(
          dbuffer, bpitch, *cpy_dbuffer, *cpitch, dmask, mpitch,
          fallback_dbuffer, fpitch, width, height);

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err);
      }
  }

  void update_bg_model(std::byte* dbuffer,
                       size_t bpitch,
                       std::byte** bg_model,
                       size_t* bg_pitch,
                       std::byte* dmask,
                       size_t mpitch,
                       const frame_info* buffer_info,
                       bool is_median)
  {
    static std::byte* dbuffer_samples[BG_NUMBER_FRAMES];
    static size_t pitches[BG_NUMBER_FRAMES];
    static int dbuffers_amount = 0;
    static double last_timestamp = 0.0;

    int height = buffer_info->height;
    int width = buffer_info->width;

    // First frame is set to the background model
    if (dbuffers_amount == 0)
      {
        // Copy buffer
        std::byte* cpy_buffer = nullptr;
        size_t cpy_pitch;
        copy_buffer(dbuffer, bpitch, &cpy_buffer, &cpy_pitch, nullptr, 0,
                    nullptr, 0, buffer_info);

        dbuffer_samples[0] = cpy_buffer;
        pitches[0] = cpy_pitch;
        dbuffers_amount = 1;
        last_timestamp = buffer_info->timestamp;

        // First bg_model is set to the buffer pointer
        // so we set it to null to reallocate new memory after
        *bg_model = nullptr;
      }
    else if (buffer_info->timestamp - last_timestamp >= BG_SAMPLING_RATE)
      {
        if (dbuffers_amount < BG_NUMBER_FRAMES)
          {
            // Copy buffer and apply mask to remove foreground
            std::byte* cpy_buffer = nullptr;
            size_t cpy_pitch;
            copy_buffer(dbuffer, bpitch, &cpy_buffer, &cpy_pitch, dmask, mpitch,
                        *bg_model, *bg_pitch, buffer_info);

            dbuffer_samples[dbuffers_amount] = cpy_buffer;
            pitches[dbuffers_amount] = cpy_pitch;
            dbuffers_amount += 1;
            last_timestamp = buffer_info->timestamp;
          }
        else
          {
            // Copy buffer on the oldest frame w/o reallocating
            // And apply mask to remove foreground
            std::byte* cpy_buffer = dbuffer_samples[0];
            size_t cpy_pitch = pitches[0];
            copy_buffer(dbuffer, bpitch, &cpy_buffer, &cpy_pitch, dmask, mpitch,
                        *bg_model, *bg_pitch, buffer_info);

            // Shift frame samples
            for (int i = 0; i < BG_NUMBER_FRAMES - 1; ++i)
              {
                dbuffer_samples[i] = dbuffer_samples[i + 1];
                pitches[i] = pitches[i + 1];
              }

            dbuffer_samples[BG_NUMBER_FRAMES - 1] = cpy_buffer;
            pitches[BG_NUMBER_FRAMES - 1] = cpy_pitch;
            last_timestamp = buffer_info->timestamp;
          }
      }
    else
      {
        return;
      }

    // Allocate device memory for background model if not already allocated
    if (*bg_model == nullptr)
      {
        hipError_t err;
        err = hipMallocPitch(bg_model, bg_pitch, width * N_CHANNELS, height);
        CHECK_CUDA_ERROR(err);
      }

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);

    // Convert samples and pitches to device memory
    std::byte** dbuffers;
    hipMalloc(&dbuffers, dbuffers_amount * sizeof(std::byte*));
    hipMemcpy(dbuffers, dbuffer_samples, dbuffers_amount * sizeof(std::byte*),
               hipMemcpyHostToDevice);
    size_t* dpitches;
    hipMalloc(&dpitches, dbuffers_amount * sizeof(size_t));
    hipMemcpy(dpitches, pitches, dbuffers_amount * sizeof(size_t),
               hipMemcpyHostToDevice);

// Estimate background
#define _BE_FPARAMS                                                            \
  dbuffers, dpitches, dbuffers_amount, *bg_model, *bg_pitch, width, height
    is_median ? estimate_background_median<<<gridSize, blockSize>>>(_BE_FPARAMS)
              : estimate_background_mean<<<gridSize, blockSize>>>(_BE_FPARAMS);
#undef _BE_FPARAMS

    hipError_t err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    hipFree(dbuffers);
    hipFree(dpitches);
  }
} // namespace

extern "C"
{
  void filter_impl(uint8_t* src_buffer,
                   const frame_info* buffer_info,
                   int th_low,
                   int th_high)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;
    int src_stride = buffer_info->stride;

    assert(N_CHANNELS == buffer_info->pixel_stride);
    std::byte *dmask, *dbuffer;
    size_t mpitch, bpitch;

    hipError_t err;

    // Allocate memory on the device
    err = hipMallocPitch(&dmask, &mpitch, width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);
    err = hipMallocPitch(&dbuffer, &bpitch, width * N_CHANNELS, height);
    CHECK_CUDA_ERROR(err);

    // Copy the input buffer to the device
    err = hipMemcpy2D(dmask, mpitch, src_buffer, src_stride,
                       width * N_CHANNELS, height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);
    err = hipMemcpy2D(dbuffer, bpitch, src_buffer, src_stride,
                       width * N_CHANNELS, height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    // Set thread block and grid dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Set first frame as bg model
    static std::byte* bg_buffer = dbuffer;
    static size_t bg_pitch = bpitch;

    // Convert RGB to LAB
    rgb_to_lab_cuda(bg_buffer, bg_pitch, dmask, mpitch, buffer_info);

    // Apply morphological opening
    opening_impl_inplace(dmask, mpitch, buffer_info);

    // Apply hysteresis thresholding
    apply_hysteresis_threshold(dmask, mpitch, buffer_info, th_low, th_high);

    // Apply masking
    apply_masking<<<gridSize, blockSize>>>(dbuffer, bpitch, dmask, mpitch,
                                           width, height);
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    // Update background model
    update_bg_model(dbuffer, bpitch, &bg_buffer, &bg_pitch, dmask, mpitch,
                    buffer_info, true);

    // Copy the result back to the host
    err = hipMemcpy2D(src_buffer, src_stride, dbuffer, bpitch,
                       width * N_CHANNELS, height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    hipFree(dmask);
    hipFree(dbuffer);
  }
}
