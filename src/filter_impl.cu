#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <cstdio>
#include <thread>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err,
           const char* const func,
           const char* const file,
           const int line)
{
  if (err != hipSuccess)
    {
      std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
      std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
      // We don't exit when we encounter CUDA errors in this example.
      std::exit(EXIT_FAILURE);
    }
}

__constant__ uint8_t* logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer
/// @param width
/// @param height
/// @param stride
/// @param pixel_stride
/// @return
__global__ void
remove_red_channel_inp(std::byte* buffer, int width, int height, int stride)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= width || y >= height)
    return;

  rgb* lineptr = (rgb*)(buffer + y * stride);
  if (y < logo_height && x < logo_width)
    {
      float alpha = logo[y * logo_width + x] / 255.f;
      lineptr[x].r = 0;
      lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1 - alpha) * 255);
      lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1 - alpha) * 255);
    }
  else
    {
      lineptr[x].r = 0;
    }
}

//******************************************************
//**                                                  **
//**               Background Estimation              **
//**                                                  **
//******************************************************

#define _BE_FSIGN                                                              \
  std::byte **buffers, size_t *bpitches, int buffers_amount, std::byte *out,   \
    size_t opitch, int width, int height

__global__ void estimate_background_mean(_BE_FSIGN)
{
#define _BACKGROUND_ESTIMATION_MEAN_SPST // single position single thread

  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;

#ifdef _BACKGROUND_ESTIMATION_MEAN_SPST
  // compute sum per channel
  int sums[N_CHANNELS] = {0};
  std::byte* ptr;
  for (int ii = 0; ii < buffers_amount; ++ii)
    {
      ptr = buffers[ii] + yy * bpitches[ii] + xx * PIXEL_STRIDE;
      for (int jj = 0; jj < N_CHANNELS; ++jj)
        sums[jj] += (int)ptr[jj];
    }

  // compute mean per channel
  ptr = out + yy * opitch + xx * PIXEL_STRIDE;
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    ptr[ii] = (std::byte)(sums[ii] / buffers_amount);
#else
#endif

#undef _BACKGROUND_ESTIMATION_MEAN_SPST
}

__device__ void _insertion_sort(std::byte* arr, int start, int end, int step)
{
  for (int ii = start + step; ii < end; ii += step)
    {
      int jj = ii;

      while (jj > start && arr[jj - step] > arr[jj])
        {
          std::byte tmp = arr[jj - step];
          arr[jj - step] = arr[jj];
          arr[jj] = tmp;
          jj -= step;
        }
    }
}

__global__ void estimate_background_median(_BE_FSIGN)
{
#define _BACKGROUND_ESTIMATION_MEDIAN_SPST // single position single thread

  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = blockIdx.x * blockDim.x + threadIdx.x;

  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;

#ifdef _BACKGROUND_ESTIMATION_MEDIAN_SPST
  // 3 channels, at most 42 buffers
  // 4 channels, at most 32 buffers
  std::byte B[128];

  // for each buffer, store pixel at (yy, xx)
  for (int ii = 0; ii < buffers_amount; ++ii)
    {
      std::byte* ptr = buffers[ii] + yy * bpitches[ii] + xx * PIXEL_STRIDE;
      int jj = ii * N_CHANNELS;
      for (int kk = 0; kk < N_CHANNELS; ++kk)
        B[jj + kk] = ptr[kk];
    }

  // the median is computed for each channel
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    _insertion_sort(B, ii, buffers_amount * N_CHANNELS, N_CHANNELS);

  // select mid
  // not treating differently even and odd `buffers_amount`
  // in order to avoid if clause inside a kernel
  std::byte* ptr = out + yy * opitch + xx * PIXEL_STRIDE;
  for (int ii = 0; ii < N_CHANNELS; ++ii)
    ptr[ii] = B[(buffers_amount / 2) * N_CHANNELS + ii];
#else
#endif

#undef _BACKGROUND_ESTIMATION_MEDIAN_SPST
}

#undef _BE_FSIGN

//******************************************************
//**                                                  **
//**                Apply Masking                     **
//**                                                  **
//******************************************************

__global__ void apply_masking(std::byte* buffer,
                              size_t bpitch,
                              std::byte* mask,
                              size_t mpitch,
                              int width,
                              int height)
{
  int xx = blockIdx.x * blockDim.x + threadIdx.x;
  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if (xx >= width || yy >= height)
    return;

  constexpr size_t PIXEL_STRIDE = N_CHANNELS;
  rgb* ipptr = (rgb*)(buffer + yy * bpitch + xx * PIXEL_STRIDE);
  rgb* mpptr = (rgb*)(mask + yy * mpitch + xx * PIXEL_STRIDE);
  int red = (int)ipptr->r + (mpptr->r > 0) * ipptr->r / 2;
  ipptr->r = (uint8_t)(red > 0xff ? 0xff : red);
}

namespace
{
  void load_logo()
  {
    static auto buffer =
      std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree};

    if (buffer == nullptr)
      {
        hipError_t err;
        std::byte* ptr;
        err = hipMalloc(&ptr, logo_width * logo_height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy(ptr, logo_data, logo_width * logo_height,
                         hipMemcpyHostToDevice);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
        CHECK_CUDA_ERROR(err);

        buffer.reset(ptr);
      }
  }
} // namespace

extern "C"
{
  void filter_impl(uint8_t* src_buffer,
                   const frame_info* buffer_info,
                   int th_low,
                   int th_high)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;
    int src_stride = buffer_info->stride;

    load_logo();

    assert(sizeof(rgb) == buffer_info->pixel_stride);
    std::byte* dBuffer;
    size_t pitch;

    hipError_t err;

    err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
    CHECK_CUDA_ERROR(err);

    err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);

    remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height,
                                                    pitch);

    err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    hipFree(dBuffer);

    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    {
      using namespace std::chrono_literals;
      //std::this_thread::sleep_for(100ms);
    }
  }
}
