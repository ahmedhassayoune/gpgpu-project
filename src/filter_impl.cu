#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <cstdio>
#include <thread>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err,
           const char* const func,
           const char* const file,
           const int line)
{
  if (err != hipSuccess)
    {
      std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
      std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
      // We don't exit when we encounter CUDA errors in this example.
      std::exit(EXIT_FAILURE);
    }
}

struct rgb
{
  uint8_t r, g, b;
};

__constant__ uint8_t* logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer
/// @param width
/// @param height
/// @param stride
/// @param pixel_stride
/// @return
__global__ void
remove_red_channel_inp(std::byte* buffer, int width, int height, int stride)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= width || y >= height)
    return;

  rgb* lineptr = (rgb*)(buffer + y * stride);
  if (y < logo_height && x < logo_width)
    {
      float alpha = logo[y * logo_width + x] / 255.f;
      lineptr[x].r = 0;
      lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1 - alpha) * 255);
      lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1 - alpha) * 255);
    }
  else
    {
      lineptr[x].r = 0;
    }
}

namespace
{
  void load_logo()
  {
    static auto buffer =
      std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree};

    if (buffer == nullptr)
      {
        hipError_t err;
        std::byte* ptr;
        err = hipMalloc(&ptr, logo_width * logo_height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy(ptr, logo_data, logo_width * logo_height,
                         hipMemcpyHostToDevice);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
        CHECK_CUDA_ERROR(err);

        buffer.reset(ptr);
      }
  }
} // namespace

extern "C"
{
  void filter_impl(uint8_t* src_buffer,
                   int width,
                   int height,
                   int src_stride,
                   int pixel_stride)
  {
    load_logo();

    assert(sizeof(rgb) == pixel_stride);
    std::byte* dBuffer;
    size_t pitch;

    hipError_t err;

    err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
    CHECK_CUDA_ERROR(err);

    err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);

    remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height,
                                                    pitch);

    err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    hipFree(dBuffer);

    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    {
      using namespace std::chrono_literals;
      //std::this_thread::sleep_for(100ms);
    }
  }
}

//******************************************************
//**                                                  **
//**           Conversion from RGB to LAB (GPU)       **
//**                                                  **
//******************************************************

#include <hip/hip_runtime.h> 
#include <iostream>
#include <cmath>
#include <algorithm>

struct rgb {
    uint8_t r, g, b;
};

struct LAB {
    float l, a, b;
};

__device__ void rgbToXyz(float r, float g, float b, float& x, float& y, float& z) {
    const float D65_XYZ[9] = {0.412453f, 0.357580f, 0.180423f,
                              0.212671f, 0.715160f, 0.072169f,
                              0.019334f, 0.119193f, 0.950227f};

    r = r / 255.0f;
    g = g / 255.0f;
    b = b / 255.0f;

    r = (r > 0.04045f) ? powf((r + 0.055f) / 1.055f, 2.4f) : (r / 12.92f);
    g = (g > 0.04045f) ? powf((g + 0.055f) / 1.055f, 2.4f) : (g / 12.92f);
    b = (b > 0.04045f) ? powf((b + 0.055f) / 1.055f, 2.4f) : (b / 12.92f);

    x = r * D65_XYZ[0] + g * D65_XYZ[1] + b * D65_XYZ[2];
    y = r * D65_XYZ[3] + g * D65_XYZ[4] + b * D65_XYZ[5];
    z = r * D65_XYZ[6] + g * D65_XYZ[7] + b * D65_XYZ[8];
}

__device__ void xyzToLab(float x, float y, float z, float& l, float& a, float& b) {
    const float D65_Xn = 0.95047f;
    const float D65_Yn = 1.00000f;
    const float D65_Zn = 1.08883f;

    x /= D65_Xn;
    y /= D65_Yn;
    z /= D65_Zn;

    float fx = (x > 0.008856f) ? powf(x, 1.0f / 3.0f) : (7.787f * x + 16.0f / 116.0f);
    float fy = (y > 0.008856f) ? powf(y, 1.0f / 3.0f) : (7.787f * y + 16.0f / 116.0f);
    float fz = (z > 0.008856f) ? powf(z, 1.0f / 3.0f) : (7.787f * z + 16.0f / 116.0f);

    l = (116.0f * fy) - 16.0f;
    a = 500.0f * (fx - fy);
    b = 200.0f * (fy - fz);
}

__device__ float labDistance(const LAB& lab1, const LAB& lab2) {
    return sqrtf(powf(lab1.l - lab2.l, 2) + powf(lab1.a - lab2.a, 2) + powf(lab1.b - lab2.b, 2));
}

__global__ void rgbToLabDistanceKernel(uint8_t* referenceBuffer, uint8_t* buffer, float* distanceArray,
                                       int width, int height, int stride, int pixelStride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uint8_t* lineptrReference = referenceBuffer + y * stride;
    uint8_t* lineptr = buffer + y * stride;

    rgb* pxlRef = (rgb*)(lineptrReference + x * pixelStride);
    float rRef = pxlRef->r;
    float gRef = pxlRef->g;
    float bRef = pxlRef->b;

    float XRef, YRef, ZRef;
    rgbToXyz(rRef, gRef, bRef, XRef, YRef, ZRef);

    float LRef, ARef, BRef;
    xyzToLab(XRef, YRef, ZRef, LRef, ARef, BRef);

    LAB referenceLab = {LRef, ARef, BRef};

    rgb* pxl = (rgb*)(lineptr + x * pixelStride);
    float r = pxl->r;
    float g = pxl->g;
    float b = pxl->b;

    float X, Y, Z;
    rgbToXyz(r, g, b, X, Y, Z);

    float L, A, B;
    xyzToLab(X, Y, Z, L, A, B);

    LAB currentLab = {L, A, B};

    float distance = labDistance(currentLab, referenceLab);
    distanceArray[y * width + x] = distance;
}

__global__ void normalizeAndConvertTo8bitKernel(uint8_t* buffer, float* distanceArray, float maxDistance,
                                                int width, int height, int stride, int pixelStride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uint8_t* lineptr = buffer + y * stride;
    rgb* pxl = (rgb*)(lineptr + x * pixelStride);

    float distance = distanceArray[y * width + x];
    uint8_t distance8bit = static_cast<uint8_t>(fminf(distance / maxDistance * 255.0f, 255.0f));

    pxl->r = distance8bit;
    pxl->g = distance8bit;
    pxl->b = distance8bit;
}

void rgb_to_lab_cuda(uint8_t* referenceBuffer, uint8_t* buffer, int width, int height, int stride, int pixelStride) {
    float* distanceArray;
    size_t distanceArraySize = width * height * sizeof(float);
    hipMalloc(&distanceArray, distanceArraySize);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    rgbToLabDistanceKernel<<<gridSize, blockSize>>>(referenceBuffer, buffer, distanceArray, width, height, stride, pixelStride);
    hipDeviceSynchronize();

    float* h_distanceArray = new float[width * height];
    hipMemcpy(h_distanceArray, distanceArray, distanceArraySize, hipMemcpyDeviceToHost);
    float maxDistance = 0.0f;
    for (int i = 0; i < width * height; ++i) {
        maxDistance = fmaxf(maxDistance, h_distanceArray[i]);
    }
    delete[] h_distanceArray;

    normalizeAndConvertTo8bitKernel<<<gridSize, blockSize>>>(buffer, distanceArray, maxDistance, width, height, stride, pixelStride);
    hipDeviceSynchronize();

    hipFree(distanceArray);
}
