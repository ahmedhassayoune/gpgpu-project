#include "hip/hip_runtime.h"
#include "filter_impl.h"

#include <cassert>
#include <chrono>
#include <cstdio>
#include <thread>
#include "logo.h"

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err,
           const char* const func,
           const char* const file,
           const int line)
{
  if (err != hipSuccess)
    {
      std::fprintf(stderr, "CUDA Runtime Error at: %s: %d\n", file, line);
      std::fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);
      // We don't exit when we encounter CUDA errors in this example.
      std::exit(EXIT_FAILURE);
    }
}

__constant__ uint8_t* logo;

/// @brief Black out the red channel from the video and add EPITA's logo
/// @param buffer
/// @param width
/// @param height
/// @param stride
/// @param pixel_stride
/// @return
__global__ void
remove_red_channel_inp(std::byte* buffer, int width, int height, int stride)
{
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x >= width || y >= height)
    return;

  rgb* lineptr = (rgb*)(buffer + y * stride);
  if (y < logo_height && x < logo_width)
    {
      float alpha = logo[y * logo_width + x] / 255.f;
      lineptr[x].r = 0;
      lineptr[x].g = uint8_t(alpha * lineptr[x].g + (1 - alpha) * 255);
      lineptr[x].b = uint8_t(alpha * lineptr[x].b + (1 - alpha) * 255);
    }
  else
    {
      lineptr[x].r = 0;
    }
}



//******************************************************
//**                                                  **
//**             Morphological Opening                **
//**                                                  **
//******************************************************



__global__ void morphological_erosion(uint8_t* buffer,
                                         uint8_t* output_buffer,
                                         int width,
                                         int height,
                                         int stride,
                                         int output_stride,
                                         int pixel_stride)
{
  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = (blockIdx.x * blockDim.x + threadIdx.x);

  if (xx >= width || yy >= height)
    return;

  uint8_t res0 = 0xFF;
  uint8_t res1 = 0xFF;
  uint8_t res2 = 0xFF;

  if (yy >= 3) {
    res0 = buffer[(yy - 3) * stride + xx * pixel_stride];
    res1 = buffer[(yy - 3) * stride + xx * pixel_stride + 1];
    res2 = buffer[(yy - 3) * stride + xx * pixel_stride + 2];
  }
  for (int i = yy - 2; i < yy; ++i) {
    if (i >= 0) {
      for (int j = xx - 2; j <= xx +2; j++) {
        if (j >= 0 && j < width) {
          res0 = min(res0, buffer[i * stride + j * pixel_stride ]);
          res1 = min(res1, buffer[i * stride + j * pixel_stride + 1]);
          res2 = min(res2, buffer[i * stride + j * pixel_stride + 2]);
        }
      }
    }
  }
  for (int j = xx - 3; j <= xx + 3; j++) {
    if (j >= 0 && j < width) {
        res0 = min(res0, buffer[yy * stride + j * pixel_stride ]);
        res1 = min(res1, buffer[yy * stride + j * pixel_stride + 1]);
        res2 = min(res2, buffer[yy * stride + j * pixel_stride + 2]);
    }
  }
  for (int i = yy + 1; i <= yy + 2; ++i) {
    if (i < width) {
      for (int j = xx - 2; j <= xx +2; j++) {
        if (j >= 0 && j < width) {
          res0 = min(res0, buffer[i * stride + j * pixel_stride ]);
          res1 = min(res1, buffer[i * stride + j * pixel_stride + 1]);
          res2 = min(res2, buffer[i * stride + j * pixel_stride + 2]);
        }
      }
    }
  }
  if (yy + 3 < width) {
    res0 = min(res0, buffer[(yy - 3) * stride + xx * pixel_stride ]);
    res1 = min(res1, buffer[(yy - 3) * stride + xx * pixel_stride + 1]);
    res2 = min(res2, buffer[(yy - 3) * stride + xx * pixel_stride + 2]);
  }

  output_buffer[yy * output_stride + xx * pixel_stride] = res0;
  output_buffer[yy * output_stride + xx * pixel_stride + 1] = res1;
  output_buffer[yy * output_stride + xx * pixel_stride + 2] = res2;
}

__global__ void morphological_dilation(uint8_t* buffer,
                                         uint8_t* output_buffer,
                                         int width,
                                         int height,
                                         int stride,
                                         int output_stride,
                                         int pixel_stride)
{
  int yy = blockIdx.y * blockDim.y + threadIdx.y;
  int xx = (blockIdx.x * blockDim.x + threadIdx.x);

  if (xx >= width || yy >= height)
    return;

  uint8_t res0 = 0x00;
  uint8_t res1 = 0x00;
  uint8_t res2 = 0x00;

  if (yy >= 3) {
    res0 = buffer[(yy - 3) * stride + xx * pixel_stride];
    res1 = buffer[(yy - 3) * stride + xx * pixel_stride + 1];
    res2 = buffer[(yy - 3) * stride + xx * pixel_stride + 2];
  }
  for (int i = yy - 2; i < yy; ++i) {
    if (i >= 0) {
      for (int j = xx - 2; j <= xx +2; j++) {
        if (j >= 0 && j < width) {
          res0 = max(res0, buffer[i * stride + j * pixel_stride ]);
          res1 = max(res1, buffer[i * stride + j * pixel_stride + 1]);
          res2 = max(res2, buffer[i * stride + j * pixel_stride + 2]);
        }
      }
    }
  }
  for (int j = xx - 3; j <= xx + 3; j++) {
    if (j >= 0 && j < width) {
        res0 = max(res0, buffer[yy * stride + j * pixel_stride ]);
        res1 = max(res1, buffer[yy * stride + j * pixel_stride + 1]);
        res2 = max(res2, buffer[yy * stride + j * pixel_stride + 2]);
    }
  }
  for (int i = yy + 1; i <= yy + 2; ++i) {
    if (i < width) {
      for (int j = xx - 2; j <= xx +2; j++) {
        if (j >= 0 && j < width) {
          res0 = max(res0, buffer[i * stride + j * pixel_stride ]);
          res1 = max(res1, buffer[i * stride + j * pixel_stride + 1]);
          res2 = max(res2, buffer[i * stride + j * pixel_stride + 2]);
        }
      }
    }
  }
  if (yy + 3 < width) {
    res0 = max(res0, buffer[(yy - 3) * stride + xx * pixel_stride ]);
    res1 = max(res1, buffer[(yy - 3) * stride + xx * pixel_stride + 1]);
    res2 = max(res2, buffer[(yy - 3) * stride + xx * pixel_stride + 2]);
  }

  output_buffer[yy * output_stride + xx * pixel_stride] = res0;
  output_buffer[yy * output_stride + xx * pixel_stride + 1] = res1;
  output_buffer[yy * output_stride + xx * pixel_stride + 2] = res2;
}

namespace
{
  void load_logo()
  {
    static auto buffer =
      std::unique_ptr<std::byte, decltype(&hipFree)>{nullptr, &hipFree};

    if (buffer == nullptr)
      {
        hipError_t err;
        std::byte* ptr;
        err = hipMalloc(&ptr, logo_width * logo_height);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpy(ptr, logo_data, logo_width * logo_height,
                         hipMemcpyHostToDevice);
        CHECK_CUDA_ERROR(err);

        err = hipMemcpyToSymbol(HIP_SYMBOL(logo), &ptr, sizeof(ptr));
        CHECK_CUDA_ERROR(err);

        buffer.reset(ptr);
      }
  }
} // namespace

extern "C"
{
  void filter_impl(uint8_t* src_buffer,
                   const frame_info* buffer_info,
                   int th_low,
                   int th_high)
  {
    int width = buffer_info->width;
    int height = buffer_info->height;
    int src_stride = buffer_info->stride;

    load_logo();

    assert(sizeof(rgb) == buffer_info->pixel_stride);
    std::byte* dBuffer;
    size_t pitch;

    hipError_t err;

    err = hipMallocPitch(&dBuffer, &pitch, width * sizeof(rgb), height);
    CHECK_CUDA_ERROR(err);

    err = hipMemcpy2D(dBuffer, pitch, src_buffer, src_stride,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);

    remove_red_channel_inp<<<gridSize, blockSize>>>(dBuffer, width, height,
                                                    pitch);

    err = hipMemcpy2D(src_buffer, src_stride, dBuffer, pitch,
                       width * sizeof(rgb), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(err);

    hipFree(dBuffer);

    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err);

    {
      using namespace std::chrono_literals;
      //std::this_thread::sleep_for(100ms);
    }
  }

  void opening_impl_inplace(uint8_t* buffer,
                          int width,
                          int height,
                          int stride,
                          int pixel_stride)
  {
      uint8_t *gpu_image;
      size_t gpu_pitch;
      hipError_t err = hipMallocPitch(&gpu_image, &gpu_pitch, width * pixel_stride * sizeof(uint8_t), height);
      CHECK_CUDA_ERROR(err);

      err = hipMemcpy2D(gpu_image, gpu_pitch, buffer, stride,
                        width * pixel_stride * sizeof(uint8_t), height, hipMemcpyDeviceToHost );
      CHECK_CUDA_ERROR(err);

      uint8_t *gpu_intermediate_image;
      size_t gpu_intermediate_pitch;
      err = hipMallocPitch(&gpu_intermediate_image, &gpu_intermediate_pitch, width * pixel_stride * sizeof(uint8_t), height);
      CHECK_CUDA_ERROR(err);

      dim3 blockSize(16, 16);
      dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x,
                  (height + (blockSize.y - 1)) / blockSize.y);
      
      morphological_erosion<<<gridSize, blockSize>>>(gpu_image,
                            gpu_intermediate_image,
                            width,
                            height,
                            gpu_pitch,
                            gpu_intermediate_pitch,
                            pixel_stride);
      err = hipDeviceSynchronize();
      CHECK_CUDA_ERROR(err);
      
      morphological_dilation<<<gridSize, blockSize>>>(gpu_intermediate_image,
                            gpu_image,
                            width,
                            height,
                            gpu_intermediate_pitch,
                            gpu_pitch,
                            pixel_stride);
      err = hipDeviceSynchronize();
      CHECK_CUDA_ERROR(err);

      err = hipMemcpy2D(buffer, stride, gpu_image, gpu_pitch,
                        width * pixel_stride * sizeof(uint8_t), height, hipMemcpyHostToDevice );
      CHECK_CUDA_ERROR(err);

  }



}
